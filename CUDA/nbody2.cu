#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>
#include <>
#define SOFTENING 1e-9f
#define dt (0.01f)
#define blocksize 256
#define dup 8
typedef struct {
	float x, y, z, vx, vy, vz;
} Body;
void randomizeBodies(float *data, int n) {
	for (int i = 0; i < n; i++) {
		data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
	}
}
__global__ void bodyForce(Body *p,int n, int numtiles) {
	register int i=threadIdx.x+blockIdx.x*blocksize,st=i/n;
	if(i>=dup*n)return;
	i%=n;
	float3 mypos={p[i].x, p[i].y, p[i].z};
	float3 acc={0.0f,0.0f,0.0f}; 
	const int partsize=blocksize/dup;
	
//	__shared__ float3 sp[blocsize];
//	for (int tile = 0, pos = 0; tile < numtiles/dup; tile++, pos += blockDim.x * dup)
//	{
//		sp[threadIdx.x].x = p[pos + st + dup * threadIdx.x].x;
//		sp[threadIdx.x].y = p[pos + st + dup * threadIdx.x].y;
//		sp[threadIdx.x].z = p[pos + st + dup * threadIdx.x].z;
//		__syncthreads();//Ensure all data are written 
//		#pragma unroll
//		for(int j = 0; j < blocksize; j++)
//		{

	__shared__ float3 sp[partsize];
	for (int tile = 0, pos = 0; tile < numtiles; tile++, pos += blockDim.x)
	{
		if(threadIdx.x%dup==st)
		{
			sp[threadIdx.x/dup].x = p[pos + threadIdx.x].x;
			sp[threadIdx.x/dup].y = p[pos + threadIdx.x].y;
			sp[threadIdx.x/dup].z = p[pos + threadIdx.x].z;
		}
		__syncthreads();//Ensure all data are written 
		#pragma unroll
		for(int j = 0; j < blocksize/dup; j++)
		{
			float3 r={0.0f,0.0f,0.0f};
			r.x=sp[j].x-mypos.x;
			r.y=sp[j].y-mypos.y;
			r.z=sp[j].z-mypos.z;
			float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + SOFTENING;
			float invDist = rsqrtf(distSqr);
			float invDist3 = invDist * invDist * invDist;
			acc.x+=r.x*invDist3;
			acc.y+=r.y*invDist3; 
			acc.z+=r.z*invDist3;
		} 
		__syncthreads();//Prevent writing before others read 
	}
	atomicAdd(&p[i].vx,acc.x*dt);
	atomicAdd(&p[i].vy,acc.y*dt);
	atomicAdd(&p[i].vz,acc.z*dt);
}

__global__ void integrate_position(Body *p,int n)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	if(i>=n)return;
		p[i].x+=p[i].vx*dt;
		p[i].y+=p[i].vy*dt;
		p[i].z+=p[i].vz*dt;
}

int main(const int argc, const char** argv)
{
	int nBodies = 1<<12;
	int salt = 0;
	if (argc > 1) nBodies = 2<<atoi(argv[1]);
	if (argc > 2) salt = atoi(argv[2]);
	const int nIters = 10;
	int blocknum=(nBodies+blocksize-1)/blocksize;
	int tilenum=(nBodies+blocksize-1)/blocksize;
	int bytes = nBodies * sizeof(Body);
	float *buf,*d_buf;
	hipHostMalloc((void**)&buf,bytes);
	hipMalloc((void**)&d_buf,bytes);
	Body *d_p,*p;
	p=(Body*)buf;
	d_p=(Body*)d_buf;
	randomizeBodies(buf, 6 * nBodies); // Init pos / vel data
	double totalTime = 0.0;
	hipMemcpy(d_buf,buf,bytes,hipMemcpyHostToDevice);
	for (int iter = 0, now = 0; iter < nIters; iter++) {
		StartTimer();
		bodyForce<<<blocknum*dup,blocksize>>>(d_p,nBodies,tilenum);
		integrate_position<<<blocknum,blocksize>>>(d_p,nBodies); 
		if(iter+1==nIters)hipMemcpy(buf,d_buf,bytes,hipMemcpyDeviceToHost);
		else hipDeviceSynchronize(); 
		const double tElapsed = GetTimer() / 1000.0;
		totalTime += tElapsed;
	}
	double avgTime = totalTime / (double)(nIters);
	float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
#ifdef ASSESS
	checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
	checkAccuracy(buf, nBodies);
	printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
	salt += 1;
#endif
	hipFree(buf);
	hipFree(d_buf);
}

